#include "RSgpu.h"
#include "RSgpu.cuh"

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

void CudaTestInf(int blocks, int threads)
{
	int targetDevice = 0;
	struct hipDeviceProp_t deviceProp;

	checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));


	testEmptyKernel << <blocks, threads >> >();

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	checkCudaErrors(hipDeviceReset());

	return;
}

bool RSgpu_CalcPressureField(
	gpureal * p_Re, gpureal * p_Im, gpureal kr, gpureal alpha_nepers,
	gpureal * xpoints, int dimx, gpureal * ypoints, int dimy, gpureal * zpoints, int dimz,
	gpureal * u_real, gpureal * u_imag, gpureal * coefficients,
	gpureal * ux, gpureal * uy, gpureal * uz,
	gpureal * unormalX, gpureal * unormalY, gpureal * unormalZ,
	int Nells, size_t numBlocks)

{

	size_t threadsPerBlock;

	int targetDevice = 0;
	struct hipDeviceProp_t deviceProp;

	checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));

	size_t globalMem = (size_t)deviceProp.totalGlobalMem;

	int globalMaxThreads = deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor;
	int numVoxelsToCompute = dimx*dimy*dimz;

	unsigned long threadToVoxWrap = 0;
	unsigned long numVoxPerKernel = 0;


	// Due to hidden memory constraints within each thread ( trig functions, FLOPs, etc.),
	// number blocks and threads manually set to a value that works.

	threadsPerBlock = THREADS_PER_BLOCK;
	//numBlocks = globalMaxThreads / threadsPerBlock;

	if (numBlocks == 0)
	{
		numBlocks = numVoxelsToCompute / threadsPerBlock + 1;

		if (Nells > 256) {
			numBlocks = 512;
		}
		if (Nells >= 1024) {
			numBlocks = 256;
		}
		if (Nells >= 2048) {
			numBlocks = 64;
		}
	}

	numVoxPerKernel = 1+ numVoxelsToCompute / (numBlocks*threadsPerBlock);
	
	if (numVoxPerKernel == 1)
		threadToVoxWrap = 0;
	else
		threadToVoxWrap = numBlocks*threadsPerBlock;

	
	std::cout << "#blocks = " << numBlocks << ", #threads per block = " << threadsPerBlock << std::endl;
	std::cout << "voxels to compute per kernel: " << numVoxPerKernel << std::endl;
	std::cout << "thread-to-voxel wrapping: " << threadToVoxWrap << std::endl;
	

	//Compute the amount memory needed for full computation
	size_t typeSize = sizeof(gpureal);
	size_t computeGridSize = dimx*dimy*dimz*typeSize;
	size_t transducerPosSize = 3 * Nells*typeSize;
	size_t transducerVecSize = 3 * Nells*typeSize;
	size_t transducerEncSize = 2 * Nells*typeSize;
	size_t coeffiecientSize = Nells*typeSize;
	size_t transducerDefSize = transducerPosSize + transducerVecSize + transducerEncSize + coeffiecientSize;

	//size_t availableMem = (globalMem - transducerDefSize);
	//std::cout << "Compute grid size = " << computeGridSize / 1024 << " kb " << std::endl;
	//std::cout << "Available mem size = " << availableMem / 1024 << " kb " << std::endl;

	//std::cout << "Allocating GPU memory..." << std::endl;
	gpureal *d_xp, *d_yp, *d_zp, *d_coeff, *d_uRe, *d_uIm, *d_pRe, *d_pIm, *d_uX, *d_uY, *d_uZ, *d_uvX, *d_uvY, *d_uvZ;

	//Allocate
	checkCudaErrors(hipMalloc((void **)&d_xp, dimx * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_yp, dimy * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_zp, dimz * typeSize));

	checkCudaErrors(hipMalloc((void **)&d_coeff, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uRe, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uIm, Nells * typeSize));
	
	checkCudaErrors(hipMalloc((void **)&d_pRe, computeGridSize ));
	checkCudaErrors(hipMalloc((void **)&d_pIm, computeGridSize ));

	checkCudaErrors(hipMalloc((void **)&d_uX, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uY, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uZ, Nells * typeSize));

	checkCudaErrors(hipMalloc((void **)&d_uvX, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uvY, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uvZ, Nells * typeSize));

	//Copy 
	checkCudaErrors(hipMemcpy(d_xp, xpoints, dimx * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_yp, ypoints, dimy * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_zp, zpoints, dimz * typeSize, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_coeff, coefficients, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uRe, u_real, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uIm, u_imag, Nells * typeSize, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_pRe, p_Re, computeGridSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_pIm, p_Im, computeGridSize, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_uX, ux, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uY, uy, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uZ, uz, Nells * typeSize, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_uvX, unormalX, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uvY, unormalY, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uvZ, unormalZ, Nells * typeSize, hipMemcpyHostToDevice));



	//std::cout << "Launching kernel..." << std::endl;
	
	unsigned long w = 0;
	unsigned long offset = 0;


	for (w = 0; w < numVoxPerKernel; w++)
	{
		std::cout << "RS Voxel chunk " << w + 1 << " / " << numVoxPerKernel << std::endl;
		CalculatePressureExpandMeshKernel << < numBlocks, threadsPerBlock >> > (
			d_pRe, d_pIm, kr, alpha_nepers,
			d_xp, dimx, d_yp, dimy, d_zp, dimz,
			d_uRe, d_uIm, d_coeff,
			d_uX, d_uY, d_uZ,
			d_uvX, d_uvY, d_uvZ, Nells, offset
			);

		checkCudaErrors(hipGetLastError());
		// Wait for the kernels to complete
		checkCudaErrors(hipDeviceSynchronize());

		offset += threadToVoxWrap;
	}
	

	//checkCudaErrors(hipDeviceSynchronize());
	//Copy result
	
	//std::cout << "Copying output to host memory..." << std::endl;
	checkCudaErrors(hipMemcpy(p_Re, d_pRe, computeGridSize, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(p_Im, d_pIm, computeGridSize, hipMemcpyDeviceToHost));

	//std::cout << "Complete" << std::endl;

	//d_xp, d_yp, d_zp, d_coeff, d_uRe, d_uIm, d_pRe, d_pIm, d_uX, d_uY, d_uZ, d_uvX, d_uvY, d_uvZ
	checkCudaErrors(hipFree(d_xp));
	checkCudaErrors(hipFree(d_yp));
	checkCudaErrors(hipFree(d_zp));
	checkCudaErrors(hipFree(d_coeff));
	checkCudaErrors(hipFree(d_uRe));
	checkCudaErrors(hipFree(d_uIm));
	checkCudaErrors(hipFree(d_pRe));
	checkCudaErrors(hipFree(d_pIm));
	checkCudaErrors(hipFree(d_uX));
	checkCudaErrors(hipFree(d_uY));
	checkCudaErrors(hipFree(d_uZ));
	checkCudaErrors(hipFree(d_uvX));
	checkCudaErrors(hipFree(d_uvY));
	checkCudaErrors(hipFree(d_uvZ));
	
	//checkCudaErrors(hipDeviceReset());

	return true;
}



bool RSgpu_CalcPressurePoints(
	gpureal * p_Re, gpureal * p_Im, gpureal kr, gpureal alpha_nepers,
	gpureal * loc_x, gpureal * loc_y, gpureal * loc_z, int nlocs,
	gpureal * u_real, gpureal * u_imag, gpureal * coefficients,
	gpureal * ux, gpureal * uy, gpureal * uz,
	gpureal * unormalX, gpureal * unormalY, gpureal * unormalZ,
	int Nells, size_t numBlocks)
{

	size_t threadsPerBlock;

	int targetDevice = 0;
	struct hipDeviceProp_t deviceProp;

	checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));

	size_t globalMem = (size_t)deviceProp.totalGlobalMem;

	int globalMaxThreads = deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor;
	int numVoxelsToCompute = nlocs;

	unsigned long threadToVoxWrap = 0;
	unsigned long numVoxPerKernel = 0;


	// Due to hidden memory constraints within each thread ( trig functions, FLOPs, etc.),
	// number blocks and threads manually set to a value that works.

	threadsPerBlock = THREADS_PER_BLOCK;
	//numBlocks = globalMaxThreads / threadsPerBlock;

	if (numBlocks == 0)
	{
		numBlocks = numVoxelsToCompute / threadsPerBlock + 1;

		if (Nells > 256) {
			numBlocks = 512;
		}
		if (Nells >= 1024) {
			numBlocks = 256;
		}
		if (Nells >= 2048) {
			numBlocks = 64;
		}
	}

	numVoxPerKernel = 1 + numVoxelsToCompute / (numBlocks*threadsPerBlock);

	if (numVoxPerKernel == 1)
		threadToVoxWrap = 0;
	else
		threadToVoxWrap = numBlocks*threadsPerBlock;


	std::cout << "#blocks = " << numBlocks << ", #threads per block = " << threadsPerBlock << std::endl;
	std::cout << "voxels to compute per kernel: " << numVoxPerKernel << std::endl;
	std::cout << "thread-to-voxel wrapping: " << threadToVoxWrap << std::endl;


	//Compute the amount memory needed for full computation
	size_t typeSize = sizeof(gpureal);
	size_t computeGridSize = nlocs*typeSize;
	size_t transducerPosSize = 3 * Nells*typeSize;
	size_t transducerVecSize = 3 * Nells*typeSize;
	size_t transducerEncSize = 2 * Nells*typeSize;
	size_t coeffiecientSize = Nells*typeSize;
	size_t transducerDefSize = transducerPosSize + transducerVecSize + transducerEncSize + coeffiecientSize;

	//size_t availableMem = (globalMem - transducerDefSize);
	//std::cout << "Compute grid size = " << computeGridSize / 1024 << " kb " << std::endl;
	//std::cout << "Available mem size = " << availableMem / 1024 << " kb " << std::endl;

	//std::cout << "Allocating GPU memory..." << std::endl;
	gpureal *d_xp, *d_yp, *d_zp, *d_coeff, *d_uRe, *d_uIm, *d_pRe, *d_pIm, *d_uX, *d_uY, *d_uZ, *d_uvX, *d_uvY, *d_uvZ;

	//Allocate
	checkCudaErrors(hipMalloc((void **)&d_xp, computeGridSize));
	checkCudaErrors(hipMalloc((void **)&d_yp, computeGridSize));
	checkCudaErrors(hipMalloc((void **)&d_zp, computeGridSize));

	checkCudaErrors(hipMalloc((void **)&d_coeff, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uRe, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uIm, Nells * typeSize));

	checkCudaErrors(hipMalloc((void **)&d_pRe, computeGridSize));
	checkCudaErrors(hipMalloc((void **)&d_pIm, computeGridSize));

	checkCudaErrors(hipMalloc((void **)&d_uX, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uY, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uZ, Nells * typeSize));

	checkCudaErrors(hipMalloc((void **)&d_uvX, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uvY, Nells * typeSize));
	checkCudaErrors(hipMalloc((void **)&d_uvZ, Nells * typeSize));

	//Copy 
	checkCudaErrors(hipMemcpy(d_xp, loc_x, computeGridSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_yp, loc_y, computeGridSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_zp, loc_z, computeGridSize, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_coeff, coefficients, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uRe, u_real, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uIm, u_imag, Nells * typeSize, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_pRe, p_Re, computeGridSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_pIm, p_Im, computeGridSize, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_uX, ux, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uY, uy, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uZ, uz, Nells * typeSize, hipMemcpyHostToDevice));

	checkCudaErrors(hipMemcpy(d_uvX, unormalX, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uvY, unormalY, Nells * typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_uvZ, unormalZ, Nells * typeSize, hipMemcpyHostToDevice));



	//std::cout << "Launching kernel..." << std::endl;

	unsigned long w = 0;
	unsigned long offset = 0;


	for (w = 0; w < numVoxPerKernel; w++)
	{
		std::cout << "RS Voxel chunk " << w + 1 << " / " << numVoxPerKernel << std::endl;
		CalculatePressureKernel << < numBlocks, threadsPerBlock >> > (
			d_pRe, d_pIm, kr, alpha_nepers,
			d_xp, d_yp, d_zp, nlocs,
			d_uRe, d_uIm, d_coeff,
			d_uX, d_uY, d_uZ,
			d_uvX, d_uvY, d_uvZ, Nells, offset
			);

		checkCudaErrors(hipGetLastError());
		// Wait for the kernels to complete
		checkCudaErrors(hipDeviceSynchronize());

		offset += threadToVoxWrap;
	}


	//checkCudaErrors(hipDeviceSynchronize());
	//Copy result

	//std::cout << "Copying output to host memory..." << std::endl;
	checkCudaErrors(hipMemcpy(p_Re, d_pRe, computeGridSize, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(p_Im, d_pIm, computeGridSize, hipMemcpyDeviceToHost));

	//std::cout << "Complete" << std::endl;

	//d_xp, d_yp, d_zp, d_coeff, d_uRe, d_uIm, d_pRe, d_pIm, d_uX, d_uY, d_uZ, d_uvX, d_uvY, d_uvZ
	checkCudaErrors(hipFree(d_xp));
	checkCudaErrors(hipFree(d_yp));
	checkCudaErrors(hipFree(d_zp));
	checkCudaErrors(hipFree(d_coeff));
	checkCudaErrors(hipFree(d_uRe));
	checkCudaErrors(hipFree(d_uIm));
	checkCudaErrors(hipFree(d_pRe));
	checkCudaErrors(hipFree(d_pIm));
	checkCudaErrors(hipFree(d_uX));
	checkCudaErrors(hipFree(d_uY));
	checkCudaErrors(hipFree(d_uZ));
	checkCudaErrors(hipFree(d_uvX));
	checkCudaErrors(hipFree(d_uvY));
	checkCudaErrors(hipFree(d_uvZ));

	//checkCudaErrors(hipDeviceReset());

	return true;
}












memsize_t getDeviceGlobalMemSize(int targetDevice )
{
	size_t            memsize = 0;

	// Query target device for maximum memory allocation
	printf(" hipGetDeviceProperties\n");
	struct hipDeviceProp_t deviceProp;
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));

	return deviceProp.totalGlobalMem;

}