

#include "BioheatCUDA.h"
#include "BioheatCUDA.cuh"

template<typename pbhe_t>
int Pennes_2ndOrder_template(pbhe_t * temp4D, pbhe_t * tdot3D, pbhe_t * kt3D, pbhe_t * rhoCp3D,
	pbhe_t * Dtxyz, pbhe_t Tblood, pbhe_t perfRate, 
	int nt,
	int nx, int ny, int nz, int bcMode=FD_FREEFLOW_BOUNDARYCOND)
{
	
	const size_t wx = 8; // # size in block x 
	const size_t wy = 4;
	const size_t warpsize = wx*wy; //should be wx*wy = 32
	// invocation should have block dims = (scale*wx,scale*wy,0) where scale^2*wx*wy = (#threadsPerBlock) and scale is an int >= 1
	// dimY = (#threadsPerBlock)/(dimX)
	// and grid dims = ( 1 + ny /  blockDim.x, 1 + nz /  blockDim.y , nx) and 
	size_t blockSize = THREADS_PER_BLOCK;
	size_t scale = 4;
	size_t blockX = wx * scale;
	size_t blockY = wy * scale;
	size_t sharedMemArrDims = (blockX + 2)*(blockY + 2);

	std::cout << "scale = " << scale << std::endl;
	std::cout << "sharedMemArrSize = " << sharedMemArrDims << std::endl;
	while ((blockX*blockY >blockSize || MAX_NUMELLS_SHARED_ARRAY < sharedMemArrDims) && scale>1) {
		
		scale--;
		std::cout << "Adjusting scale to " << scale << std::endl;
		blockX = wx * scale;
		blockY = wy * scale;
		sharedMemArrDims = (blockX + 2)*(blockY + 2);
		std::cout << "sharedMemArrSize = " << sharedMemArrDims << std::endl;
		std::cout << "block X,Y = " << blockX << ", " << blockY << std::endl;
	}

	if (MAX_NUMELLS_SHARED_ARRAY < sharedMemArrDims) {
		std::cout << "Error, not enough space for shared arrays. Adjust either 'MAX_NUMELLS_SHARED_ARRAY' or 'THREADS_PER_BLOCK' and re-compile" << std::endl;
		return -1;
	}
	
	dim3 griddims3(1 + ny / blockX, 1 + nz / blockY, nx);
	dim3 blockdims(blockX, blockY, 1);
	
	pbhe_t * temp3D_tf_dev, *temp3D_ti_dev, *tdot3D_dev, *kt3D_dev, *rhoCp3D_dev, *tmp, * Dtxyz_d;

	std::cout << "Block dimensions: (" << blockdims.x << ", " << blockdims.y << ", " << blockdims.z << ")"<< std::endl;
	std::cout << " Grid dimensions: (" << griddims3.x << ", " << griddims3.y << ", " << griddims3.z << ")" << std::endl;
	int targetDevice = 0;
	struct hipDeviceProp_t deviceProp; 
	
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, targetDevice));

	// Allocate
	size_t typeSize = sizeof(pbhe_t);

	checkCudaErrors(hipMalloc((void **)&temp3D_tf_dev, nx *ny*nz* typeSize));
	checkCudaErrors(hipMalloc((void **)&temp3D_ti_dev, nx *ny*nz * typeSize));
	checkCudaErrors(hipMalloc((void **)&tdot3D_dev, nx *ny*nz * typeSize));
	checkCudaErrors(hipMalloc((void **)&kt3D_dev, nx *ny*nz * typeSize));
	checkCudaErrors(hipMalloc((void **)&rhoCp3D_dev, nx *ny*nz * typeSize));
	checkCudaErrors(hipMalloc((void **)&Dtxyz_d, 4 * typeSize));

	checkCudaErrors(hipMemcpy(temp3D_ti_dev, temp4D, nx *ny*nz*  typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(tdot3D_dev, tdot3D, nx *ny*nz*  typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(kt3D_dev, kt3D, nx *ny*nz*  typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(rhoCp3D_dev, rhoCp3D, nx *ny*nz* typeSize, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(Dtxyz_d, Dtxyz, 4* typeSize, hipMemcpyHostToDevice));

	//int * numwrites_d;
	//size_t intsize = sizeof(int);
	//checkCudaErrors(hipMalloc((void **)&numwrites_d, nx *ny*nz * intsize));
	//checkCudaErrors(hipMemcpy(numwrites_d, numwrites, nx *ny*nz*  intsize, hipMemcpyHostToDevice));

	pbhe_t * T_in, *T_out;
	T_in = temp3D_ti_dev;
	T_out = temp3D_tf_dev;
	int ti;
	for (ti = 0; ti < nt - 1; ti++)
	{
		std::cout << "ti = " << ti << std::endl;
		Pennes_2ndOrder_cuda_kernel<pbhe_t> << < griddims3, blockdims >> > (T_out, T_in, tdot3D_dev, kt3D_dev, rhoCp3D_dev, Dtxyz_d, Tblood, perfRate, nx, ny, nz, bcMode);
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipMemcpy( (temp4D + (ti+1)*nx*ny*nz ) , T_out, nx *ny*nz* typeSize, hipMemcpyDeviceToHost));
		//checkCudaErrors(hipMemcpy((numwrites + (ti+1)*nx*ny*nz), numwrites_d, nx *ny*nz*intsize, hipMemcpyDeviceToHost));
		tmp = T_in;
		T_in = T_out;
		T_out = tmp ;

	}

	checkCudaErrors(hipGetLastError());
	// Wait for the kernels to complete
	checkCudaErrors(hipDeviceSynchronize());

	//checkCudaErrors(hipFree(numwrites_d));

	checkCudaErrors(hipFree(temp3D_tf_dev));
	checkCudaErrors(hipFree(temp3D_ti_dev));
	checkCudaErrors(hipFree(tdot3D_dev));
	checkCudaErrors(hipFree(kt3D_dev));
	checkCudaErrors(hipFree(rhoCp3D_dev));
	checkCudaErrors(hipFree(Dtxyz_d));

	checkCudaErrors(hipDeviceReset());




	return 0;
};

int Pennes_2ndOrder_GPU(double * temp4D, double * tdot3D, double * kt3D, double * rhoCp3D,
	double * Dtxyz, double Tblood, double perfRate,
	int nt,
	int nx, int ny, int nz, int bcMode)
{
	return Pennes_2ndOrder_template<double>(temp4D, tdot3D, kt3D, rhoCp3D, Dtxyz, Tblood, perfRate, nt, nx, ny, nz, bcMode);
}


int Pennes_2ndOrder_GPU_f(float * temp4D, float * tdot3D, float * kt3D, float * rhoCp3D,
	float * Dtxyz, float Tblood, float perfRate,
	int nt,
	int nx, int ny, int nz, int bcMode)
{
	return Pennes_2ndOrder_template<float>(temp4D, tdot3D, kt3D, rhoCp3D, Dtxyz, Tblood, perfRate, nt, nx, ny, nz, bcMode);
}